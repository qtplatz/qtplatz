/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "cudacolormap.hpp"
#include <adportable/debug.hpp>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime_api.h>
#include <math.h>

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/device_new.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <memory>
#include <thread>
#include <mutex>

namespace cuda {

    struct Color {
        float r; float g; float b; float v;
        Color( float _r = 0, float _g = 0, float _b = 0, float _v = 0 ) : r(_r), g(_g), b(_b), v(_v) {}
        float blue() const { return b; }
        float green() const { return g; }
        float red() const { return r; }
        float value() const { return v; }
    };

    __constant__ float __levels[] = { 0.0, 0.2, 0.4, 0.6, 0.8, 0.97, 1.0 }; // 7 steps

    class ColorMap {
        thrust::device_vector< Color > colors_;
    public:
        ColorMap() {
            colors_.push_back( Color( 0,     0, 0.0, 0.00 ) );
            colors_.push_back( Color( 0,     0, 0.5, 0.20 ) );
            colors_.push_back( Color( 0,   1.0, 1.0, 0.40 ) ); // cyan
            colors_.push_back( Color( 0,   1.0,   0, 0.60 ) ); // green
            colors_.push_back( Color( 1.0, 1.0,   0, 0.80 ) ); // yellow
            colors_.push_back( Color( 1.0,   0,   0, 0.97 ) ); // red
            colors_.push_back( Color( 1.0, 1.0, 1.0, 1.00 ) ); // white
        }


        __device__ const Color color( float value ) const {

            thrust::device_vector< Color > results(1);
            
            auto it
                = thrust::lower_bound( colors_.begin(), colors_.end()
                                       , &value, &value
                                       , results.begin()
                                       , []( const Color& c, const float& v )->bool{
                                           return c.value < v;
                                       } );
                                 
#if 0
            thrust::device_vector< Color > results(1);
            thrust::device_vector< float > values;
            values.push_back( value );

            ColorMap::const_iterator it
                = thrust::lower_bound( colors_.begin(), colors_.end()
                                       , values.begin(), values.end()
                                       , results.begin()
                                       , []( const Color& c, const float& v )->bool{
                                           return c.value < v;
                                       } );

            if ( it == colors_.end() )
                return Color( colors_.back() );
            
            if ( it == colors_.begin() )
                return Color( *it );

            thrust::device_reference< const Color > ref = *it;

            auto prev = it - 1;
            //ADDEBUG() << results[0]->r; // << ", " << thrust::get< 0 >( *prev );

            ColorMap::const_iterator prev = it - 1;
            float frac = ( value - prev->value ) / ( it->value - prev->value );
            
            float r = ( it->r - prev->r ) * frac + prev->r;
            float g = ( it->g - prev->g ) * frac + prev->g;
            float b = ( it->b - prev->b ) * frac + prev->b;
            
            return Color( r, g, b );
#endif            
            return Color( 0, 0, 0, 0 );
        }
    };

}

using namespace cuda;

static thrust::device_ptr< cuda::ColorMap > __colorMap__;

void
cudaApplyColorMap( const cv::Mat& src, cv::Mat& dst, float scale )
{
    static std::once_flag flag;
    std::call_once( flag, [](){ __colorMap__ = thrust::device_new< cuda::ColorMap >(); } );

    if ( src.type() != CV_32F )
        return;

    ADDEBUG() << "cudaApplycolormap";
    
    dst = cv::Mat( src.rows, src.cols, CV_8UC3 );

    for ( size_t i = 0; i < src.rows; ++i ) {
        for ( size_t j = 0; j < src.cols; ++j ) {
            float v = src.at< float >( i, j ) * scale;
            auto c = __colorMap__.color( v );
            dst.at< cv::Vec3b >( i, j )[ 0 ] = c.blue();
            dst.at< cv::Vec3b >( i, j )[ 1 ] = c.green();
            dst.at< cv::Vec3b >( i, j )[ 2 ] = c.red();
        }
    }
    return;
}

template<typename T>
struct Fun
{
    __device__ T operator()(T t1, T t2)  {
        auto result = t1+t2;
        return result;
    }
};

int
run()
{
    const int N = 100;
    thrust::device_vector<int> vec(N);
    thrust::sequence(vec.begin(),vec.end());
    auto op = Fun<int>();
    return thrust::reduce(vec.begin(),vec.end(),0,op);
}

