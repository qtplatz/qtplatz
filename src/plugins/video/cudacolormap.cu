/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "cudacolormap.hpp"
#include <adportable/debug.hpp>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime_api.h>
#include <math.h>

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/device_new.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <memory>
#include <thread>
#include <mutex>

namespace cuda {

    template<typename T>
    struct Fun
    {
        __device__ T operator()(T t1, T t2)  {
            auto result = t1+t2;
            return result;
        }
    };

    int
    run()
    {
        const int N = 100;
        thrust::device_vector<int> vec(N);
        thrust::sequence(vec.begin(),vec.end());
        auto op = Fun<int>();
        return thrust::reduce(vec.begin(),vec.end(),0,op);
    }

    struct Color {
        float r; float g; float b; float v;
        __host__ __device__ Color( float _r = 0, float _g = 0, float _b = 0, float _v = 0 ) : r(_r), g(_g), b(_b), v(_v) {}
        __host__ __device__ float blue() const { return b; }
        __host__ __device__ float green() const { return g; }
        __host__ __device__ float red() const { return r; }
        __host__ __device__ float value() const { return v; }
    };

    template< typename T > struct grater_than {
        T value_;
        __host__ __device__ grater_than( T value ) : value_( value ) {}
        __host__ __device__ bool operator ()( const Color& a ) { return a.v > value_; }
    };

    class ColorMap {
        thrust::device_vector< Color > colors_;
    public:
        __host__ __device__ ColorMap() {
            colors_.push_back( Color( 0,     0, 0.0, 0.00 ) );
            colors_.push_back( Color( 0,     0, 0.5, 0.20 ) );
            colors_.push_back( Color( 0,   1.0, 1.0, 0.40 ) ); // cyan
            colors_.push_back( Color( 0,   1.0,   0, 0.60 ) ); // green
            colors_.push_back( Color( 1.0, 1.0,   0, 0.80 ) ); // yellow
            colors_.push_back( Color( 1.0,   0,   0, 0.97 ) ); // red
            colors_.push_back( Color( 1.0, 1.0, 1.0, 1.00 ) ); // white
        }

        __host__ __device__ const Color color( float value ) const {
#if 0
            auto it = thrust::find_if( colors_.begin(), colors_.end(), grater_than<float>( value ) );

            if ( it == colors_.end() )
                return Color( colors_.back() );
            
            if ( it == colors_.begin() )
                return Color( *it );
#endif

#if 0                
            ColorMap::const_iterator it
                = thrust::lower_bound( colors_.begin(), colors_.end()
                                       , values.begin(), values.end()
                                       , results.begin()
                                       , []( const Color& c, const float& v )->bool{
                                           return c.value < v;
                                       } );


            thrust::device_reference< const Color > ref = *it;

            auto prev = it - 1;
            //ADDEBUG() << results[0]->r; // << ", " << thrust::get< 0 >( *prev );

            ColorMap::const_iterator prev = it - 1;
            float frac = ( value - prev->value ) / ( it->value - prev->value );
            
            float r = ( it->r - prev->r ) * frac + prev->r;
            float g = ( it->g - prev->g ) * frac + prev->g;
            float b = ( it->b - prev->b ) * frac + prev->b;
            
            return Color( r, g, b );
#endif            
            return Color( 0, 0, 0, 0 );
        }
    };

}

using namespace cuda;

static thrust::device_ptr< cuda::ColorMap > __colorMap__;

void
cudaApplyColorMap( const cv::Mat& src, cv::Mat& dst, float scale )
{
#if 0
    static std::once_flag flag;
    std::call_once( flag, [](){ __colorMap__ = thrust::device_new< cuda::ColorMap >(); } );

    if ( src.type() != CV_32F )
        return;
    dst = cv::Mat( src.rows, src.cols, CV_8UC3 );

    auto rptr = thrust::raw_pointer_cast( __colorMap__ );

    for ( size_t i = 0; i < src.rows; ++i ) {
        for ( size_t j = 0; j < src.cols; ++j ) {
            float v = src.at< float >( i, j ) * scale;
            auto c = rptr->color( v );
            dst.at< cv::Vec3b >( i, j )[ 0 ] = c.blue();
            dst.at< cv::Vec3b >( i, j )[ 1 ] = c.green();
            dst.at< cv::Vec3b >( i, j )[ 2 ] = c.red();
        }
    }
#endif
    return;
}

