#include "hip/hip_runtime.h"
/**************************************************************************
** Copyright (C) 2010-2017 Toshinobu Hondo, Ph.D.
** Copyright (C) 2013-2017 MS-Cheminformatics LLC, Toin, Mie Japan
*
** Contact: toshi.hondo@qtplatz.com
**
** Commercial Usage
**
** Licensees holding valid MS-Cheminfomatics commercial licenses may use this file in
** accordance with the MS-Cheminformatics Commercial License Agreement provided with
** the Software or, alternatively, in accordance with the terms contained in
** a written agreement between you and MS-Cheminformatics.
**
** GNU Lesser General Public License Usage
**
** Alternatively, this file may be used under the terms of the GNU Lesser
** General Public License version 2.1 as published by the Free Software
** Foundation and appearing in the file LICENSE.TXT included in the
** packaging of this file.  Please review the following information to
** ensure the GNU Lesser General Public License version 2.1 requirements
** will be met: http://www.gnu.org/licenses/old-licenses/lgpl-2.1.html.
**
**************************************************************************/

#include "colormap.hpp"
#include "cvtypes.hpp"
#include <adportable/debug.hpp>
#include <boost/numeric/ublas/matrix.hpp>
#include <QImage>

namespace cuda {

    namespace rgb {
        
        enum RGB { Red = 0, Green = 1, Blue = 2 };
    
        template<typename T>
        struct cvColor {
            const size_t nlevels_;
            const T * colors_;

            template< typename U > struct Color {
                __device__ Color( U r, U g, U b ) : red(r), green(g), blue(b) {}
                U red, green, blue;
            };
            
            __device__ cvColor( size_t num, const T* rgb ) : nlevels_( num ), colors_(rgb) {
            }

            __device__ inline T R( int level ) const { return colors_[ level ]; }
            __device__ inline T G( int level ) const { return colors_[ level + nlevels_ ]; }
            __device__ inline T B( int level ) const { return colors_[ level + nlevels_ * 2 ]; }
            
            __device__ inline Color< T > operator ()( size_t level, float frac ) const {
                if ( level >= nlevels_ )
                    return Color< T > ( R( nlevels_ - 1 ), G( nlevels_ - 1 ), B( nlevels_ - 1 ) );
                if ( level == 0 )
                    return Color< T > ( R( 0 ), G( 0 ), B( 0 ) );
                auto prev = level - 1;
                return Color< T > ( ( ( R( level ) - R( prev ) ) * frac + R( prev ) )
                                    , ( ( G( level ) - G( prev ) ) * frac + G( prev ) )
                                    , ( ( B( level ) - B( prev ) ) * frac + B( prev ) ) );
            }
        };
    } // rgb

} // namespace cuda


template<typename T > __global__ void
colormap_kernel( const int num, const T * d_x, uint8_t * d_y
                 , const int nlevels, const float * d_levels, const float * d_colors
                 , float scaleFactor
                 , bool isBGR = false )
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    if ( id < num ) {

        using namespace cuda::rgb;

        cvColor<float> cvColor( nlevels, d_colors );    
        
        float r( cvColor.R(0) ), g( cvColor.G(0) ), b( cvColor.B( 0 ) );
        float frac(0);
        
        int level = 0;
        T dx = scaleFactor * d_x[ id ];

        while ( level < nlevels ) {
            if ( dx < d_levels[ level ] )
                break;
            ++level;
        }
        if ( level > 0 )
            frac = ( dx - d_levels[ level - 1 ] ) / ( d_levels[ level ] - d_levels[ level - 1 ] );

        auto c = cvColor( level, frac );

        if ( isBGR ) {
            d_y[(id * 3) + 0] = c.blue * 255;
            d_y[(id * 3) + 1] = c.green * 255;
            d_y[(id * 3) + 2] = c.red * 255;
        } else {
            d_y[(id * 3) + 0] = c.red * 255;
            d_y[(id * 3) + 1] = c.green * 255;
            d_y[(id * 3) + 2] = c.blue * 255;
        }
    }
}

namespace cuda {

    struct ColorMapHelper {

        const thrust::device_vector< float >& d_levels_;
        const thrust::device_vector< float >& d_colors_;

        ColorMapHelper( const thrust::device_vector< float >& levels
                        , const thrust::device_vector< float >& colors ) : d_levels_( levels )
                                                                         , d_colors_( colors ) {
        }

        // matrix<> -> apply colormap --> QImage
        template< typename T > QImage operator()( const boost::numeric::ublas::matrix< T >& m, double scaleFactor ) const {
            
            const int num = m.size1() * m.size2();
            const int threads = 64; // 1024; // 512; //256;
            const int blocks = (num / threads) + ((num % threads) ? 1 : 0 );
            
            //auto p_m = reinterpret_cast< const T * >( m.data() );
            auto p_m = m.data().begin();
            
            T * d_m(0);
            hipMalloc( &d_m, num * sizeof( T ) );
            hipMemcpyAsync( d_m, p_m, num * sizeof( T ), hipMemcpyHostToDevice );
            
            uint8_t * d_rgb(0);
            hipMalloc( &d_rgb, num * 3 * sizeof(uint8_t) );

            colormap_kernel <<< blocks, threads >>>
                ( num
                  , d_m
                  , d_rgb
                  , d_levels_.size()
                  , thrust::raw_pointer_cast( d_levels_.data() )
                  , thrust::raw_pointer_cast( d_colors_.data() )
                  , float( scaleFactor )
                  , false // isBGR ?
                    );
            //----------
            QImage rgb( m.size1(), m.size2(), QImage::Format_RGB888 );    

            hipMemcpyAsync( rgb.bits(), d_rgb, num * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );

            hipFree( d_m );
            hipFree( d_rgb );

            hipStreamSynchronize( 0 );    
            return rgb;                
        }

        // cv::Mat -> apply color map --> QImage
        QImage operator()( const cv::Mat& gray, double scaleFactor ) const {
            const int num = gray.cols * gray.rows;
            const int threads = 64; // 1024; // 512; //256;
            const int blocks = (num / threads) + ((num % threads) ? 1 : 0 );
    
            const float * p_gray = reinterpret_cast< const float * >( gray.ptr() );

            //thrust::device_vector< float > d_gray( p_gray, p_gray + num );    
            float * d_gray(0);
            hipMalloc( &d_gray, num * sizeof( float ) );
            hipMemcpyAsync( d_gray, p_gray, num * sizeof( float ), hipMemcpyHostToDevice );

            uint8_t * d_rgb(0);
            hipMalloc( &d_rgb, num * 3 * sizeof(uint8_t) );

            colormap_kernel <<< blocks, threads >>>
                ( num
                  , d_gray
                  , d_rgb
                  , d_levels_.size()
                  , thrust::raw_pointer_cast( d_levels_.data() )
                  , thrust::raw_pointer_cast( d_colors_.data() )
                  , float( scaleFactor )                  
                  , true
                    );    
            //--
            QImage rgb( gray.rows, gray.cols, QImage::Format_RGB888 );    

            hipMemcpyAsync( rgb.bits(), d_rgb, num * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );

            hipFree( d_gray );
            hipFree( d_rgb );
            
            hipStreamSynchronize( 0 );
            return rgb;
        }
    };
}

///////////////////////

cuda::ColorMap::ColorMap( const std::vector< float >& levels
                          , const std::vector< float >& colors )
    : d_levels_( levels.begin(), levels.end() )
    , d_colors_( colors.begin(), colors.end() )
{
}

cuda::ColorMap::~ColorMap()
{
}

template<>
QImage
cuda::ColorMap::operator()( const boost::numeric::ublas::matrix<float>& m, double scaleFactor ) const
{
    return cuda::ColorMapHelper( d_levels_, d_colors_ )( m, scaleFactor );
}

template<>
QImage
cuda::ColorMap::operator()( const boost::numeric::ublas::matrix<double>& m, double scaleFactor ) const
{
    return cuda::ColorMapHelper( d_levels_, d_colors_ )( m, scaleFactor );
}

#if HAVE_OPENCV
QImage
cuda::ColorMap::operator()( const cv::Mat& gray, double scaleFactor ) const
{
    return cuda::ColorMapHelper( d_levels_, d_colors_ )( gray, scaleFactor );
}
#endif
