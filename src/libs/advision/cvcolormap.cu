#include "hip/hip_runtime.h"
/**************************************************************************
** Copyright (C) 2010-2017 Toshinobu Hondo, Ph.D.
** Copyright (C) 2013-2017 MS-Cheminformatics LLC, Toin, Mie Japan
*
** Contact: toshi.hondo@qtplatz.com
**
** Commercial Usage
**
** Licensees holding valid MS-Cheminfomatics commercial licenses may use this file in
** accordance with the MS-Cheminformatics Commercial License Agreement provided with
** the Software or, alternatively, in accordance with the terms contained in
** a written agreement between you and MS-Cheminformatics.
**
** GNU Lesser General Public License Usage
**
** Alternatively, this file may be used under the terms of the GNU Lesser
** General Public License version 2.1 as published by the Free Software
** Foundation and appearing in the file LICENSE.TXT included in the
** packaging of this file.  Please review the following information to
** ensure the GNU Lesser General Public License version 2.1 requirements
** will be met: http://www.gnu.org/licenses/old-licenses/lgpl-2.1.html.
**
**************************************************************************/

#include "cvcolormap.hpp"
#include "cvtypes.hpp"

namespace cuda {

    namespace bgr {

        enum RGB { Red = 0, Green = 1, Blue = 2 };
    
        template<typename T>
        struct cvColor {
            const size_t nlevels_;
            const T * colors_;

            template< typename U > struct Color {
                __device__ Color( U r, U g, U b ) : red(r), green(g), blue(b) {}
                U red, green, blue;
            };
            
            __device__ cvColor( size_t num, const T* rgb ) : nlevels_( num ), colors_(rgb) {
            }

            __device__ inline T R( int level ) const { return colors_[ level ]; }
            __device__ inline T G( int level ) const { return colors_[ level + nlevels_ ]; }
            __device__ inline T B( int level ) const { return colors_[ level + nlevels_ * 2 ]; }
            
            __device__ inline Color< T > operator ()( size_t level, float frac ) const {
                if ( level >= nlevels_ )
                    return Color< T > ( R( nlevels_ - 1 ), G( nlevels_ - 1 ), B( nlevels_ - 1 ) );
                if ( level == 0 )
                    return Color< T > ( R( 0 ), G( 0 ), B( 0 ) );
                auto prev = level - 1;
                return Color< T > ( ( ( R( level ) - R( prev ) ) * frac + R( prev ) )
                                    , ( ( G( level ) - G( prev ) ) * frac + G( prev ) )
                                    , ( ( B( level ) - B( prev ) ) * frac + B( prev ) ) );
            }
        };
    } // bgr

} // namespace cuda

__global__
void
cv_colormap_kernel( const int num, const float * d_x, uint8_t * d_y
                    , const int nlevels, const float * d_levels, const float * d_colors )
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    if ( id < num ) {

        using namespace cuda::bgr;

        cvColor<float> cvColor( nlevels, d_colors );    
        
        float r( cvColor.R(0) ), g( cvColor.G(0) ), b( cvColor.B( 0 ) );
        float frac(0);
        
        int level = 0;

        while ( level < nlevels ) {
            if ( d_x[ id ] < d_levels[ level ] )
                break;
            ++level;
        }
        if ( level > 0 )
            frac = ( d_x[ id ] - d_levels[ level - 1 ] ) / ( d_levels[ level ] - d_levels[ level - 1 ] );

        auto c = cvColor( level, frac );

        // BGR packed
        d_y[(id * 3) + 0] = c.blue * 255;
        d_y[(id * 3) + 1] = c.green * 255;
        d_y[(id * 3) + 2] = c.red * 255;
    }
}

///////////////////////

cuda::cvColorMap::cvColorMap( const std::vector< float >& levels
                              , const std::vector< float >& colors )
    : d_levels_( levels.begin(), levels.end() )
    , d_colors_( colors.begin(), colors.end() )
{
}

cuda::cvColorMap::~cvColorMap()
{
}

cv::Mat
cuda::cvColorMap::operator()( const cv::Mat& gray ) const
{
    const int num = gray.cols * gray.rows;
    const int threads = 64; // 1024; // 512; //256;
    const int blocks = (num / threads) + ((num % threads) ? 1 : 0 );
    
    const float * p_gray = reinterpret_cast< const float * >( gray.ptr() );

    //thrust::device_vector< float > d_gray( p_gray, p_gray + num );    
    float * d_gray(0);
    hipMalloc( &d_gray, num * sizeof( float ) );
    hipMemcpyAsync( d_gray, p_gray, num * sizeof( float ), hipMemcpyHostToDevice );

    // thrust::device_vector< unsigned char > d_rgb( num * 3 );  // row major array, rgb
    uint8_t * d_rgb(0);
    hipMalloc( &d_rgb, num * 3 * sizeof(uint8_t) );
#if 0
    cv_colormap_kernel <<< blocks, threads >>>
        ( num
          , thrust::raw_pointer_cast( d_gray.data() )
          , thrust::raw_pointer_cast( d_rgb.data() )
          , d_levels_.size()
          , thrust::raw_pointer_cast( d_levels_.data() )
          , thrust::raw_pointer_cast( d_colors_.data() )
            );
#else
    cv_colormap_kernel <<< blocks, threads >>>
        ( num
          , d_gray
          , d_rgb
          , d_levels_.size()
          , thrust::raw_pointer_cast( d_levels_.data() )
          , thrust::raw_pointer_cast( d_colors_.data() )
            );    
#endif
    
    cv::Mat rgb( gray.rows, gray.cols, CV_8UC(3) ); // BGR

    // thrust::copy( d_rgb.begin(), d_rgb.end(), rgb.ptr() );
    hipMemcpyAsync( rgb.ptr(), d_rgb, num * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost );

    hipFree( d_gray );
    hipFree( d_rgb );
    
    // hipDeviceSynchronize();
    hipStreamSynchronize( 0 );

    return rgb;
}
