#include "hip/hip_runtime.h"
/**************************************************************************
** Copyright (C) 2010-2017 Toshinobu Hondo, Ph.D.
** Copyright (C) 2013-2017 MS-Cheminformatics LLC, Toin, Mie Japan
*
** Contact: toshi.hondo@qtplatz.com
**
** Commercial Usage
**
** Licensees holding valid MS-Cheminfomatics commercial licenses may use this file in
** accordance with the MS-Cheminformatics Commercial License Agreement provided with
** the Software or, alternatively, in accordance with the terms contained in
** a written agreement between you and MS-Cheminformatics.
**
** GNU Lesser General Public License Usage
**
** Alternatively, this file may be used under the terms of the GNU Lesser
** General Public License version 2.1 as published by the Free Software
** Foundation and appearing in the file LICENSE.TXT included in the
** packaging of this file.  Please review the following information to
** ensure the GNU Lesser General Public License version 2.1 requirements
** will be met: http://www.gnu.org/licenses/old-licenses/lgpl-2.1.html.
**
**************************************************************************/

#include "aftypes.hpp"
#include <arrayfire.h>
#include <af/hip/hip_runtime.h>

namespace advision { namespace af {

        enum RGB { Red = 0, Green = 1, Blue = 2 };
    
        template<typename T>
        struct afColor {
            const size_t nlevels_;
            const T * colors_;

            template< typename U > struct Color {
                __device__ Color( U r, U g, U b ) : red(r), green(g), blue(b) {}
                U red, green, blue;
            };
            
            __device__ afColor( size_t num, const T* rgb ) : nlevels_( num ), colors_(rgb) {
            }

            __device__ inline T R( int level ) const { return colors_[ level ]; }
            __device__ inline T G( int level ) const { return colors_[ level + nlevels_ ]; }
            __device__ inline T B( int level ) const { return colors_[ level + nlevels_ * 2 ]; }
            
            __device__ inline Color< T > operator ()( size_t level, float frac ) const {
                if ( level >= nlevels_ )
                    return Color< T > ( R( nlevels_ - 1 ), G( nlevels_ - 1 ), B( nlevels_ - 1 ) );
                if ( level == 0 )
                    return Color< T > ( R( 0 ), G( 0 ), B( 0 ) );
                auto prev = level - 1;
                return Color< T > ( ( ( R( level ) - R( prev ) ) * frac + R( prev ) )
                                    , ( ( G( level ) - G( prev ) ) * frac + G( prev ) )
                                    , ( ( B( level ) - B( prev ) ) * frac + B( prev ) ) );
            }
        };

    } // namespace af
} // namespace advision


__global__
void
af_colormap_kernel( const int num, const float * d_x, uint8_t * d_y
                    , const int nlevels, const float * d_levels, const float * d_colors )
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    if ( id < num ) {

        using namespace advision::af;

        afColor<float> afColor( nlevels, d_colors );    
        
        float r( afColor.R(0) ), g( afColor.G(0) ), b( afColor.B( 0 ) );
        float frac(0);
        
        int level = 0;

        while ( level < nlevels ) {
            if ( d_x[ id ] < d_levels[ level ] )
                break;
            ++level;
        }
        if ( level > 0 )
            frac = ( d_x[ id ] - d_levels[ level - 1 ] ) / ( d_levels[ level ] - d_levels[ level - 1 ] );

        auto c = afColor( level, frac );
        
        d_y[id + num * 0] = c.blue * 255;
        d_y[id + num * 1] = c.green * 255;
        d_y[id + num * 2] = c.red * 255;
    }
}

af::array
afColorMap( const af::array& gray, const af::array& levels, const af::array& colors )
{
    // Ensure any JIT kernels have executed
    gray.eval();
    levels.eval();
    colors.eval();

    // Determine ArrayFire's CUDA stream
    int cuda_id = afcu::getNativeId( af::getDevice() );
    hipStream_t af_cuda_stream = afcu::getStream( cuda_id );

    const int num = gray.dims(0) * gray.dims(1);

    const float * d_gray = gray.device< float >();

    using advision::af_type_value;

    // result array
    af::array rgb = af::constant< uint8_t >( 0, gray.dims(0), gray.dims(1), 3, af_type_value< uint8_t >::value );

    uint8_t * d_rgb = rgb.device< uint8_t >();
    const float * d_levels = levels.device< float >();
    const float * d_colors = colors.device< float >();

    const int threads = 256;
    const int blocks = (num / threads) + ((num % threads) ? 1 : 0 );

    af_colormap_kernel <<< blocks, threads, 0, af_cuda_stream >>> ( num, d_gray, d_rgb, levels.dims(0), d_levels, d_colors );

    hipDeviceSynchronize();

    gray.unlock();
    rgb.unlock();    
    levels.unlock();
    colors.unlock();

    return rgb;
}

