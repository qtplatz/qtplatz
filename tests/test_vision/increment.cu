#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <arrayfire.h>
# include <af/hip/hip_runtime.h>

template<typename T>
struct Fun
{
    __device__ T operator()(T t1, T t2)  {
        auto result = t1+t2;
        return result;
    }
};

__global__
void
increment_kernel( const int num, const float * d_x, float * d_y )
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    if ( id < num )
        d_y[id] = -d_x[id];
}

void
increment( af::array& a )
{
    a.eval(); // Ensure any JIT kernels have executed

    
    // Determine ArrayFire's CUDA stream
    int afid = af::getDevice();
    int cudaid = afcu::getNativeId( afid );
    hipStream_t af_cuda_stream = afcu::getStream( cudaid );

    af::array b = af::constant( 0, a.dims(0), a.dims(1), f32 );
    
    const float * d_a = a.device< float >();
    float * d_b = b.device< float >();

    increment_kernel <<< 6, 1, 0, af_cuda_stream >>>( 6, d_a, d_b );

    hipDeviceSynchronize();

    a.unlock();
    af::print( "b", b );
}

