#include "hip/hip_runtime.h"
/**************************************************************************
** Copyright (C) 2010-2017 Toshinobu Hondo, Ph.D.
** Copyright (C) 2013-2017 MS-Cheminformatics LLC, Toin, Mie Japan
*
** Contact: toshi.hondo@qtplatz.com
**
** Commercial Usage
**
** Licensees holding valid MS-Cheminfomatics commercial licenses may use this file in
** accordance with the MS-Cheminformatics Commercial License Agreement provided with
** the Software or, alternatively, in accordance with the terms contained in
** a written agreement between you and MS-Cheminformatics.
**
** GNU Lesser General Public License Usage
**
** Alternatively, this file may be used under the terms of the GNU Lesser
** General Public License version 2.1 as published by the Free Software
** Foundation and appearing in the file LICENSE.TXT included in the
** packaging of this file.  Please review the following information to
** ensure the GNU Lesser General Public License version 2.1 requirements
** will be met: http://www.gnu.org/licenses/old-licenses/lgpl-2.1.html.
**
**************************************************************************/

#include "aftypes.hpp"
#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <arrayfire.h>
#include <af/hip/hip_runtime.h>

typedef uint8_t rgb_type;

namespace af_extension {

    enum RGB { Red = 0, Green = 1, Blue = 2 };
    
    template<typename T>
    struct ColorTable {
        const int num_;
        const T * rgb_;

        __device__ ColorTable( int num, const T* rgb ) : num_( num ), rgb_(rgb) {
        }

        __device__ inline T color( int level, RGB code ) const {
            return rgb_[ level + num_ * code ];
        }

        __device__ T operator ()( int level, float frac, RGB code ) const {
            if ( level == 0 )
                return color( level, code );
            else if ( level == num_ )
                return color( num_ - 1, code );
            else if ( level > 0 )
                return ( color( level, code ) - color ( level - 1, code ) ) * frac + color( level - 1, code );
            else
                return T(0);
        }        
    };
}

__global__
void
colormap_kernel( const int num, const float * d_x, rgb_type * d_y
                 , const int nlevels, const float * d_levels, const float * d_colors )
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    float r(0), g(0), b(0), frac(0);
    int level = 0;

    af_extension::ColorTable<float> table( nlevels, d_colors );

    if ( id < num ) {
        while ( level < nlevels ) {
            if ( d_x[ id ] < d_levels[ level ] )
                break;
            ++level;
        }
        if ( level > 0 ) {
            frac = ( d_x[ id ] - d_levels[ level - 1 ] ) / ( d_levels[ level ] - d_levels[ level - 1 ] );
            r = table( level, frac, af_extension::Red  );
            g = table( level, frac, af_extension::Green );
            b = table( level, frac, af_extension::Blue );
        }

        d_y[id + num * 0] = r * 255;
        d_y[id + num * 1] = g * 255;
        d_y[id + num * 2] = b * 255;
    }
}

af::array
colorMap( const af::array& gray, const af::array& levels, const af::array& colors )
{
    gray.eval(); // Ensure any JIT kernels have executed
    levels.eval();
    colors.eval();
    int cuda_id = afcu::getNativeId( af::getDevice() ); // Determine ArrayFire's CUDA stream
    hipStream_t af_cuda_stream = afcu::getStream( cuda_id );

    const int num = gray.dims(0) * gray.dims(1);

    const float * d_gray = gray.device< float >();

    using namespace arrayfire;
    
    // result array

    af::array rgb = af::constant< rgb_type >( 0, gray.dims(0), gray.dims(1), 3, af_type_value< rgb_type >::value );
    rgb_type * d_rgb = rgb.device< rgb_type >();

    const float * d_levels = levels.device< float >();
    const float * d_colors = colors.device< float >();

    const int threads = 256;
    const int blocks = (num / threads) + ((num % threads) ? 1 : 0 );

    colormap_kernel <<< blocks, threads, 0, af_cuda_stream >>> ( num, d_gray, d_rgb, levels.dims(0), d_levels, d_colors );

    hipDeviceSynchronize();

    rgb.unlock();

    return rgb;
}

